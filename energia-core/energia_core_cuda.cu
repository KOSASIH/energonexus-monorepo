#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void energia_device_read_data_kernel(int *device_id, float *data, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        // Read data from device hardware
        data[idx] = device_id[idx] * 2.0f;
    }
}

__global__ void energia_device_write_data_kernel(int *device_id, float *data, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        // Write data to device hardware
        device_id[idx] = (int)data[idx] / 2;
    }
}

extern "C" {
    void energia_device_read_data(int *device_id, float *data, int len) {
        int blockSize = 256;
        int numBlocks = (len + blockSize - 1) / blockSize;
        energia_device_read_data_kernel<<<numBlocks, blockSize>>>(device_id, data, len);
        hipDeviceSynchronize();
    }

    void energia_device_write_data(int *device_id, float *data, int len) {
        int blockSize = 256;
        int numBlocks = (len + blockSize - 1) / blockSize;
        energia_device_write_data_kernel<<<numBlocks, blockSize>>>(device_id, data, len);
        hipDeviceSynchronize();
    }
}
